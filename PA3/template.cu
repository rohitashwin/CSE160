#include "hip/hip_runtime.h"
#include <gputk.h>

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  // Calculate the row index of the C element and A
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  // Calculate the column index of C and B
  int Col = blockIdx.x * blockDim.x + threadIdx.x;

  if ((Row < numCRows) && (Col < numCColumns)) {
    float Cvalue = 0.0;
    // each thread computes one element of the block sub-matrix
    for (int k = 0; k < numAColumns; ++k) {
      Cvalue += A[Row * numAColumns + k] * B[k * numBColumns + Col];
    }
    C[Row * numCColumns + Col] = Cvalue;
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(sizeof(float) * numCRows * numCColumns);
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  gpuTKLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  gpuTKCheck(hipMalloc((void **)&deviceA, sizeof(float) * numARows * numAColumns));
  gpuTKCheck(hipMalloc((void **)&deviceB, sizeof(float) * numBRows * numBColumns));
  gpuTKCheck(hipMalloc((void **)&deviceC, sizeof(float) * numCRows * numCColumns));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  gpuTKCheck(hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice));
  gpuTKCheck(hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice));
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid((numCColumns - 1) / 16 + 1, (numCRows - 1) / 16 + 1, 1);
  dim3 dimBlock(16, 16, 1);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows,
                               numAColumns, numBRows,
                               numBColumns, numCRows,
                               numCColumns);
  gpuTKCheck(hipGetLastError());
  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  gpuTKCheck(hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost));
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}